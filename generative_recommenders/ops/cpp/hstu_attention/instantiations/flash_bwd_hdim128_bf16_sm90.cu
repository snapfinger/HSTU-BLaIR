/* Copyright (c) Meta Platforms, Inc. and affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Copyright (c) 2024, Jay Shah, Ganesh Bikshandi, Ying Zhang, Vijay Thakkar,
// Pradeep Ramani, Tri Dao. Splitting the different template instantiations to
// different files to speed up compilation. This file is auto-generated. See
// "generate_kernels.py"

#include "flash_bwd_launch_template.h"

#ifndef FLASHATTENTION_DISABLE_HDIM128
template <>
void run_mha_bwd_<90, cutlass::bfloat16_t, 128>(
    Flash_bwd_params& params,
    hipStream_t stream) {
  run_mha_bwd_hdim128<90, cutlass::bfloat16_t>(params, stream);
}
#endif
